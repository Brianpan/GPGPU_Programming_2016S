#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

//right hand side
__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	int *neigh_t,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;

	//is in the bound
	if(yt < ht and xt < wt and mask[curt] > 127.0f){
		const int yb = oy + yt, xb = ox + xt;
		
		// add offset to background
		if(0 <= yb and yb < hb and 0 <= xb and xb < wb){
			

			// margin is background
			for(int neighbor = -1 ; neighbor <= 1; neighbor = neighbor +2){
				
				int x_tmp, y_tmp, t_x_tmp, t_y_tmp;	
				// x axis
				x_tmp = xb + neighbor;
				y_tmp = yb;
				t_x_tmp = xt + neighbor;
				t_y_tmp = yt;
				if(x_tmp >= 0 and x_tmp < wb and t_x_tmp >= 0 and t_x_tmp < wt){
					// add number of neighbor
					neigh_t[curt] += 1;
					
					// if is margin
					int n_curt = wb*y_tmp + x_tmp;
					int t_curt = t_y_tmp*wt + t_x_tmp;

					// 3 dim target
					fixed[curt*3 + 0] += target[t_curt*3 + 0]- target[curt*3 + 0];
					fixed[curt*3 + 1] += target[t_curt*3 + 1] - target[curt*3 + 1];
					fixed[curt*3 + 2] += target[t_curt*3 + 2] - target[curt*3 + 2];
					
					// if neighbor is margin add value
					if(mask[t_curt] <= 127.0f){
						for(int i = 0; i < 3; i++){
							fixed[curt*3 + i] += background[n_curt*3 + i];
						}
					}
				}
				
				// y axis
				x_tmp = xb;
				y_tmp = yb + neighbor;
				t_x_tmp = xt;
				t_y_tmp = yt + neighbor;
				if(y_tmp >=0 and y_tmp < hb and t_y_tmp >= 0 and t_y_tmp < ht){
					// add number of neighbor
					neigh_t[curt] += 1;
					
					// if is margin
					int n_curt = wb*y_tmp + x_tmp;
					int t_curt = wt*t_y_tmp + t_x_tmp;

					// 3 dim target
					fixed[curt*3 + 0] += target[t_curt*3 + 0] - target[curt*3 + 0];
					fixed[curt*3 + 1] += target[t_curt*3 + 1] - target[curt*3 + 1];
					fixed[curt*3 + 2] += target[t_curt*3 + 2] - target[curt*3 + 2];
					
					// if neighbor is margin add value
					if(mask[t_curt] <= 127.0f){
						for(int i = 0; i < 3; i++){
							fixed[curt*3 + i] += background[n_curt*3 + i];
						}
					}
				}

			}
		}
	}

	return;
}

__global__ void PoissonImageCloningIteration(
	const float *fixed, 
	const float *mask, 
	const int *neigh_t,
	float *updated,
	float *new_data, 
	const int wt, 
	const int ht
){
	const int yt = blockIdx.y*blockDim.y + threadIdx.y;
	const int xt = blockIdx.x*blockDim.x + threadIdx.x;
	const int curt = yt*wt + xt;
	


	if(xt < wt and yt < ht and mask[curt] > 127.0f){
		
		// check neighbor to form equation
		// sparse matrix
		//right hand value
		float temp[3];
		temp[0] = fixed[curt];
		temp[1] = fixed[curt + 1];
		temp[2] = fixed[curt + 2];
		
		// do b-AX
		for(int neighbor = -1 ; neighbor <= 1 ; neighbor = neighbor + 2){
			int x_tmp, y_tmp, t_cur;

			// x axis
			x_tmp = xt + neighbor;
			y_tmp = yt;
			t_cur = x_tmp + y_tmp*wt;
			
			// if is margin
			if(x_tmp >= 0 and x_tmp < wt and mask[t_cur] > 127.0f){
				for(int i = 0 ; i < 3 ; i ++){
					temp[i] += updated[3*t_cur + i];
				}
			}	
			// y axis
			x_tmp = xt;
			y_tmp = yt + neighbor;
			t_cur = x_tmp + y_tmp*wt;

			if(y_tmp >= 0 and y_tmp < ht and mask[t_cur] > 127.0f){
				for(int i = 0 ; i < 3 ; i ++){
					temp[i] += updated[3*t_cur + i];
				}
			}
		}

		for(int i = 0 ; i < 3 ; i ++){
			// minus n*current point
			temp[i] -= neigh_t[curt]*updated[3*curt + i];
			
			// save to buf
			new_data[3*curt + i] += temp[i];
		}
	}

	return;
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{	
	// poisson image editing
	float *fixed, *buf1, *buf2;
	int *neigh_t;

	hipMalloc(&neigh_t, sizeof(int)*wt*ht);
	hipMalloc(&fixed, 3*sizeof(float)*wt*ht);
	hipMalloc(&buf1, 3*sizeof(float)*wt*ht);
	hipMalloc(&buf2, 3*sizeof(float)*wt*ht);

	// initialize neighbor to 0
	hipMemset(neigh_t, 0, sizeof(int)*wt*ht);
	hipMemset(fixed, 0, 3*sizeof(float)*wt*ht);
	
	
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	// right hand side
	CalculateFixed<<<gdim, bdim>>>(background, target, mask, fixed, neigh_t,
								 wb, hb, wt, ht, oy, ox);
	hipDeviceSynchronize();

	// poisson initialized x = 0
	hipMemset(buf1, 0, 3*sizeof(float)*wt*ht);
	hipMemset(buf2, 0, 3*sizeof(float)*wt*ht);
	// hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

	// iter
	for(int i= 0 ; i < 10000; i++){
		PoissonImageCloningIteration<<<gdim, bdim>>>(
			fixed, mask, neigh_t, buf1 ,buf2, wt, ht
		);
		hipDeviceSynchronize();
		hipMemcpy(buf1, buf2, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	}

	//easy post
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);

	hipFree(neigh_t);
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
