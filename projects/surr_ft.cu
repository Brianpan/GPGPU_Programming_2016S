#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#define SIGSIZE 5
#define SIGDIM 1
#define NBLK 256
#define TIMESLOT 439
//exp i
//https://devtalk.nvidia.com/default/topic/505308/complex-number-exponential-function/
__device__ float angle_trans(const hipComplex& z){
	return atan2(hipCimagf(z), hipCrealf(z));
}

__global__ void fft_polar_angle(hipfftComplex *data, float *angle, float *mag, int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	//abs of fft
	mag[idx] = hipCabsf(data[idx]);
	//angle of fft
	angle[idx] = angle_trans(data[idx]);
	return;
}

int main(int argc, char **argv)
{	

	// host assign
	hipfftComplex *h_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex)*SIGSIZE*SIGDIM);

	for(int i = 0; i < SIGSIZE*SIGDIM; i ++){
		// h_signal[i].x = rand() / (float) RAND_MAX;
		h_signal[i].x = i;
		h_signal[i].y = 0;
	}
	int data_size = SIGSIZE*SIGDIM;
	int mem_size = sizeof(hipfftComplex)*data_size;
	hipfftComplex *d_signal;
	checkCudaErrors(hipMalloc((void **) &d_signal, mem_size));

	checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

	//cufft
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, SIGSIZE, HIPFFT_C2C, SIGDIM) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}

	//forward transform
	printf("---Transform fft--- \n");
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);

	//do angle implement in matlab
	float *d_angle, *d_mag;
	checkCudaErrors(hipMalloc(&d_angle, sizeof(float)*data_size));
	checkCudaErrors(hipMalloc(&d_mag, sizeof(float)*data_size));

	fft_polar_angle<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_angle, d_mag, data_size);

	float *h_angle = (float *) malloc(sizeof(float)*data_size);
	float *h_mag = (float *) malloc(sizeof(float)*data_size);

	checkCudaErrors(hipMemcpy(h_angle, d_angle, sizeof(float)*data_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_mag, d_mag, sizeof(float)*data_size, hipMemcpyDeviceToHost));

	// backward transform
	// printf("---Inverse fft transform --- \n");
	// hipfftExecC2C(plan, d_signal, d_signal, 
							   // HIPFFT_BACKWARD);
	
	hipfftComplex *h_inverse_signal = (hipfftComplex *) malloc(sizeof(hipfftComplex)*SIGDIM*SIGSIZE);
	checkCudaErrors(hipMemcpy(h_inverse_signal, d_signal, mem_size, hipMemcpyDeviceToHost));

	for(int i = 0; i < SIGSIZE*SIGDIM; i ++){
		if(i%SIGSIZE == 0){
			printf("---- column %d started ---- \n", i/SIGSIZE+1);
		}
		printf("before: %f , %f ; after: %f , %f ; angle: %f ; mag: %f \n",
				h_signal[i].x, h_signal[i].y,
				h_inverse_signal[i].x, h_inverse_signal[i].y, 
				h_angle[i], h_mag[i]);
	}

	free(h_signal);
	// free(h_inverse_signal);
	
	hipfftDestroy(plan);
	checkCudaErrors(hipFree(d_signal));

	hipDeviceReset();
	
	return 0;
}	