#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "Timer.h"

#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>

#define SIGSIZE 439
#define SIGDIM 10000
#define NBLK 256
#define TIMESLOT 439
#define pi 3.14159

__device__ float angle_trans(const hipComplex& z){
	return atan2(hipCimagf(z), hipCrealf(z));
}

__global__ void fft_polar_angle(hipfftComplex *data, float *angle, float *mag, int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	//abs of fft
	mag[idx] = hipCabsf(data[idx]);
	//angle of fft
	angle[idx] = angle_trans(data[idx]);
	return;
}

// do p(2:N)=[p1 -flipud(p1)];
__global__ void odd_surr_trans(float *angle, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	

	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// p(1) is not necessary for changing
	if(data_idx ==0){
		return;
	}
	
	int half_idx;
	//p(2: 2+half-1)
	if(data_idx <= half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
			
	// -flipup(p1)	
	}else{
		int diff = data_idx - half_sig_size;
		int reverse_data_idx = half_sig_size- diff;
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];
	}

	return;
}

__global__ void even_surr_trans(float *angle, float *mag, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	
	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// angle part
	// p(2:N)=[p1' p(h+1) -flipud(p1)'];
	int half_idx;
	// 0 nothing
	// 1->half_sig_size-1
	if(data_idx == 0 || data_idx == half_sig_size){
		angle[idx] = angle[idx];
	}
	else if(1<= data_idx < half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
	}
	// half_sig_size
	// half_sig_size+1->data_size-1
	if(data_idx > half_sig_size){
		int diff = data_idx - half_sig_size+1;
		int reverse_data_idx = half_sig_size- diff;
		
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];

		// magnitude part
		// m=[flipud(m(2:h))];
		diff = data_idx - (half_sig_size);
		reverse_data_idx = (half_sig_size) - diff;
		int mag_idx = reverse_data_idx + data_col*sig_size;
		mag[idx] = mag[mag_idx];
	}

	return;

}
// s(:,j)=m.*exp(i*p);
__global__ void i_mul_trans(hipfftComplex *result, const float *mag, const float *angle, int data_size){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= data_size){
		return;
	}
	float mag_val = mag[idx];
	float angle_val = angle[idx];
	result[idx].x = mag_val*cosf(angle_val);
	result[idx].y = mag_val*sinf(angle_val);

	return;
}

__global__ void get_real_trans(float *result, const hipfftComplex *data_list, const int data_size, const int sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx] = data_list[idx].x/sig_size;
	return;
}

__global__ void real2cufft_trans(hipfftComplex *result, const float *input, const int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx].x = input[idx];
	result[idx].y = 0;
	return;
}

void phaseran(float *result, const int data_num, const int time_size){
	int data_size = data_num*time_size;
	int mem_size = sizeof(hipfftComplex)*data_size;
	
	hipfftComplex *d_signal;
	checkCudaErrors(hipMalloc((void **) &d_signal, mem_size));
	float *d_input;
	checkCudaErrors(hipMalloc(&d_input, sizeof(float)*data_size));
	checkCudaErrors(hipMemcpy(d_input, result, sizeof(float)*data_size, hipMemcpyHostToDevice));
	
	real2cufft_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_input, data_size);
	hipDeviceSynchronize();

	//cufft
	hipfftHandle plan_r, plan;
	
	if (hipfftPlan1d(&plan_r, time_size, HIPFFT_R2C, data_num) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftPlan1d(&plan, time_size, HIPFFT_C2C, data_num) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}

	//forward transform
	// printf("---Transform fft--- \n");
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
	checkCudaErrors(hipFree(d_input));
	
	//do angle implement in matlab
	float *d_angle, *d_mag;
	checkCudaErrors(hipMalloc(&d_angle, sizeof(float)*data_size));
	checkCudaErrors(hipMalloc(&d_mag, sizeof(float)*data_size));

	fft_polar_angle<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_angle, d_mag, data_size);
	hipDeviceSynchronize();

	checkCudaErrors(hipFree(d_signal));

	// start parallel surrogate
	int half_col_size = time_size/2;
	int half_size = half_col_size*data_num;
	float *d_ran_series;
	
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, rand()%10000);
	
	if(time_size%2==0){
		//assign half minus 1
		int half_minus_one_size = (half_col_size-1)*data_num;
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_minus_one_size));
		hiprandGenerateUniform(gen, d_ran_series, half_minus_one_size);

		even_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_mag, d_ran_series, data_size, time_size, half_col_size);
		
	}else{		
		
		//assign half 
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_size));
		//random generator
		
		hiprandGenerateUniform(gen, d_ran_series, half_size);
		
		// do column vector trans p(2:N)=[p1 -flipud(p1)];
		odd_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_ran_series, data_size, time_size, half_col_size);
	}
	hipDeviceSynchronize();

	checkCudaErrors(hipFree(d_ran_series));
	hiprandDestroyGenerator(gen);
	
	// multiply with m.*exp(i*p) = m*cos(p) + m*i*sin(p)
	hipfftComplex *d_i_mul;
	checkCudaErrors(hipMalloc((void **) &d_i_mul, mem_size));
	
	i_mul_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_i_mul, d_mag, d_angle, data_size);
	hipDeviceSynchronize();

	// backward transform
	// printf("---Inverse fft transform --- \n");
	hipfftExecC2C(plan, d_i_mul, d_i_mul, 
							   HIPFFT_BACKWARD);
	float *d_result;
	checkCudaErrors(hipMalloc(&d_result, sizeof(float)*data_size));

	get_real_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_result, d_i_mul, data_size, time_size);
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(result, d_result, sizeof(float)*data_size, hipMemcpyDeviceToHost));
	
	hipfftDestroy(plan);
	
	checkCudaErrors(hipFree(d_angle));
	checkCudaErrors(hipFree(d_mag));
	checkCudaErrors(hipFree(d_result));
	checkCudaErrors(hipFree(d_i_mul));
	hipDeviceReset();
	return;
}

// sort by time points
// __global__ void thrust_sort(float *data, const int viewers, const int randomNum, const int timePoints){
// 	int idx = threadIdx.x + blockDim.x*blockIdx.x;
// 	if(idx >= viewers*randomNum){
// 		return;
// 	}
// 	thrust::sort(thrust::seq, data+idx*timePoints, data+(idx+1)*timePoints);	
	
// 	return;
// }
int mod_pivot = 0;
int mod_fuc(){
	return (mod_pivot++)/SIGSIZE;
}

void sortData(float *data, const int viewers, const int randomNum, const int timePoints){
	// float *d_tmp_data;
	int total_size = viewers*randomNum*timePoints;
	// checkCudaErrors(hipMalloc(&d_tmp_data, sizeof(float)*total_size));
	// checkCudaErrors(hipMemcpy(d_tmp_data, data, sizeof(float)*total_size, hipMemcpyDeviceToDevice));
	
	// thrust::device_ptr<float> data_ptr(data);
	// thrust_sort<<<(viewers*randomNum+NBLK-1)/NBLK, NBLK>>>(data, viewers, randomNum, timePoints);
	// for(int i = 0 ; i < viewers*randomNum ; i ++){
	// 	int time_size = timePoints;
	// }
	// thrust::sort(data_ptr, data_ptr+viewers*randomNum*timePoints, col_sort_functor(viewers, randomNum, data));


	// sort by key implement
	//vectorized sort
	thrust::host_vector<int> h_rank(total_size);
	thrust::generate(h_rank.begin(), h_rank.end(), mod_fuc);
  	thrust::device_vector<int> d_rank = h_rank;
  	thrust::device_ptr<float> d_result = thrust::device_pointer_cast(data);
  	thrust::device_vector<float> d_ori(d_result, d_result+total_size);
  	thrust::device_vector<float> d_v_result(d_result, d_result+total_size);
  	  
  	thrust::stable_sort_by_key(d_v_result.begin(), d_v_result.end(), d_rank.begin());
  	thrust::stable_sort_by_key(d_rank.begin(), d_rank.end(), d_v_result.begin());
  	hipDeviceSynchronize();

  	float *raw_ptr = thrust::raw_pointer_cast(d_v_result.data());
  	checkCudaErrors(hipMemcpy(data, raw_ptr, total_size*sizeof(float), hipMemcpyDeviceToDevice));
	
	// thrust::host_vector<float> h_sort = d_v_result;
	// thrust::host_vector<float> h_data = d_ori;
	// for(int i = 0;i < total_size; i++){
	// 	if(i%timePoints == 0){
	// 		printf("-------\n");
	// 	}
	// 	printf("raw: %f sorted: %f \n", h_data[i], h_sort[i]);
	// } 
	return;
}

// aaft : cudaPointer return value
// data : cudaPointer input data subjects (with several viewers)
// viewers : # of viewers in data
// randomNum : # of random series
// timePoints :  # of time slots
void amplitudeAdjustedFourierTransform(double *d_aaft, const double *d_data, const int viewers, const int randomNum, const int timePoints) {
	// generate normal random variables
	int total_size = viewers*randomNum*timePoints;
	float *d_normal;
	checkCudaErrors(hipMalloc(&d_normal, sizeof(float)*total_size));
	// generator 
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, rand()%10000);
	hiprandGenerateNormal(gen, d_normal, total_size, 0, 1);
	hiprandDestroyGenerator(gen);
	// sort d_normal
	sortData(d_normal, viewers, randomNum, timePoints);
	
	checkCudaErrors(hipFree(d_normal));
	return;
}

int main(int argc, char **argv)
{	
	//phaseran(result, SIGDIM, SIGSIZE);
	int viewers = 5;
	double *result = (double *)malloc(sizeof(double)*SIGSIZE*SIGDIM*viewers);	
	Timer phaseran_timer;
	phaseran_timer.Start();
	for(int i = 0; i <1 ; i++){

		for(int i = 0; i<viewers*SIGSIZE*SIGDIM;i++){
			// if(i%SIGSIZE==0){
			// 	printf("---data %d:----\n", i/SIGSIZE);
			// }
			result[i] = (double) rand()/RAND_MAX;
			// printf("rand: %f\n", result[i]);
		}
		double *d_result;
		hipMalloc(&d_result, sizeof(double)*viewers*SIGSIZE*SIGDIM);
		hipMemcpy(d_result, result, sizeof(double)*viewers*SIGSIZE*SIGDIM, hipMemcpyHostToDevice);
		
		double *db_result;
		hipMalloc(&db_result, sizeof(double)*viewers*SIGSIZE*SIGDIM);

		amplitudeAdjustedFourierTransform(db_result, d_result, viewers, SIGDIM, SIGSIZE);
		// sortData(d_result, viewers, SIGDIM, SIGSIZE);
		hipMemcpy(result, d_result, sizeof(double)*viewers*SIGSIZE*SIGDIM, hipMemcpyDeviceToHost);

		// for(int i = 0; i <viewers*SIGSIZE*SIGDIM;i++){
		// 	if(i%SIGSIZE==0){
		// 		printf("---!!data %d:----\n", i/SIGSIZE);
		// 	}
		// 	printf("sorted:%f \n", result[i]);
		// }
		
		hipFree(d_result);
		hipFree(db_result);
		
	}
	phaseran_timer.Pause();
	printf_timer(phaseran_timer);
	free(result);
	return 0;
}	