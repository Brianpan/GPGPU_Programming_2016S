#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#define SIGSIZE 5
#define SIGDIM 3
int main(int argc, char **argv)
{	

	// host assign
	hipfftComplex *h_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex)*SIGSIZE*SIGDIM);

	for(int i = 0; i < SIGSIZE*SIGDIM; i ++){
		h_signal[i].x = rand() / (float) RAND_MAX;
		h_signal[i].y = 0;
	}

	int mem_size = sizeof(hipfftComplex)*SIGSIZE*SIGDIM;
	hipfftComplex *d_signal;
	checkCudaErrors(hipMalloc((void **) &d_signal, mem_size));

	checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

	//cufft
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, SIGSIZE, HIPFFT_C2C, SIGDIM) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}

	//forward transform
	printf("---Transform fft--- \n");
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);

	//backward transform
	printf("---Inverse fft transform --- \n");
	hipfftExecC2C(plan, d_signal, d_signal, 
							   HIPFFT_BACKWARD);
	
	hipfftComplex *h_inverse_signal = (hipfftComplex *) malloc(sizeof(hipfftComplex)*SIGDIM*SIGSIZE);
	checkCudaErrors(hipMemcpy(h_inverse_signal, d_signal, mem_size, hipMemcpyDeviceToHost));

	for(int i = 0; i < SIGSIZE*SIGDIM; i ++){
		if(i%SIGSIZE == 0){
			printf("---- column %d started ---- \n", i/SIGSIZE+1);
		}
		printf("before: %f , %f ; after: %f , %f \n",
				h_signal[i].x, h_signal[i].y,
				h_inverse_signal[i].x/(float)SIGSIZE, h_inverse_signal[i].y/(float)SIGSIZE);
	}

	free(h_signal);
	free(h_inverse_signal);
	
	hipfftDestroy(plan);
	checkCudaErrors(hipFree(d_signal));

	hipDeviceReset();
	
	return 0;
}	